#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random/normal_distribution.h>

#include "parameters.h"
#include "setup.h"
#include "xlog.h"
//x6y

using namespace std;

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCVA[5];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){normalizedCVA[i]=0;}
	}
};

//operator to be called in thrust binary operation
__host__ __device__
counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR)
{
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++)
	{
		tempCVA.normalizedCVA[i]=cvaL.normalizedCVA[i]+cvaR.normalizedCVA[i];
	}
	return tempCVA;
}

struct get_CVA : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	__host__ __device__
	counterpartyCVA operator()(unsigned int seed)
	{
		//initialize output counterparty results
		counterpartyCVA sumCVA;

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		//thrust::uniform_real_distribution<float> u01(0,1);
		thrust::random::experimental::normal_distribution<float> ndist(0.1f, 1.0f);

		//initialize parameters for simulation
		float timeStep=YEARS/float(NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		double price=STARTING_PRICE;
		float discount=1;

		//factor used in random evolution of price
		float priceFactor=sqrt(VARIANCE)*(YEARS/float(NUM_TIMESTEPS));

		//to hold the random normal generated each step
		float normal=0;

		//initialize hazard rate factors (TO BE PARAMETRIZED?
		float hazard[5];
		for (int i=0;i<5;i++)
		{
			hazard[i]=BASE_HAZARD+BASE_HAZARD*float(i);
		}

		//run the required number of steps
		//NOTE: TO BE OPTIMIZED
		for(unsigned int i = 0; i < NUM_TIMESTEPS-1; ++i)
		{
			time=time+timeStep;
			//get new price
			normal=ndist(rng);
			price+=price*normal*priceFactor;
			//get discount for current step
			discount=1.0/exp(DISCOUNT*time);
			//find default probability for each and copy result to output CVA struct
			for (int j=0;j<5;j++)
			{
				defProb=1.0f/exp((time-timeStep)*hazard[j])-1.0f/exp(time*hazard[j]);
//				cout<<j<<" defprob: "<<defProb<<" discount: "<<discount<<" price: "<<price<<endl;
				sumCVA.normalizedCVA[j]+=defProb*discount*price;
//				cout<<i<<" type: "<<j<<" CVA norm: "<<(defProb*discount*price)<<endl;
			}
		}
		return sumCVA;
	}
};

counterpartyCVA genPaths()
{
	thrust::plus<counterpartyCVA> binary_op;
	counterpartyCVA cpCVA;
	cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA(),cpCVA,binary_op);
	for (int i=0;i<5;i++)
	{cpCVA.normalizedCVA[i]=cpCVA.normalizedCVA[i]/float(NUM_SIMULATIONS);}
	return cpCVA;
}

float getCumulativeCVA(counterpartyCVA& cpCVA,vector<counterParties>& cp)
{
	float sumCVA=0;
	int partiesFifth = PARTIES_NUM / 5;
	for (int j = 0; j < 5; j++) {
		int startCount = partiesFifth * j;
		for (long i = 0; i < partiesFifth; i++) {
			sumCVA+=cpCVA.normalizedCVA[j]*cp[startCount + i].netDeal;
		}
	}
	return sumCVA;
}

int main(){
	XLog logMain("CVA Main");
	logMain.log("Starting..");
	vector<counterParties> cp(PARTIES_NUM);
	{
		XLog logAlloc("Setup");
		setupCounterparties(cp);
		logAlloc.log("Counterparties Setup");
		allocateDeals(cp);
		logAlloc.log("Deal allocation complete");
	}

	counterpartyCVA cpCVA;
	{
		XLog logPath("Path simulation");
		cpCVA=genPaths();
	}

	float totalCVA;
	{
		XLog logSum("Sum CVA");
		totalCVA=getCumulativeCVA(cpCVA,cp);
		logSum.log("total CVA:",totalCVA);
	}

	return 0;
}


