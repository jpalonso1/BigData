#include "hip/hip_runtime.h"
#include "parameters.h"
#include "setup.h"
#include <vector>

//x6y
//#include <thrust/fill.h>
//#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
//#include <thrust/copy.h>
//#include <thrust/fill.h>
//#include <thrust/replace.h>
//#include <thrust/functional.h>

using namespace std;

struct get_CVA : public thrust::unary_function<unsigned int,float>
{
	//hazard rate
	const float hr;

	//initialize
	get_CVA(float _hr):hr(_hr){}

	__host__ __device__
	float operator()(unsigned int seed)
	{
		float sumCVA = 0;
		unsigned int N = NUM_SIMULATIONS; // samples per thread

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		thrust::uniform_real_distribution<float> u01(0,1);

		float timeStep=YEARS/float(NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		double price=STARTING_PRICE;

		float factor=sqrt(VARIANCE)*(YEARS/float(NUM_TIMESTEPS));
		float normalRandom=0;
		;
		float discount=1;
		//run the required number of steps
		for(unsigned int i = 0; i < N; ++i)
		{
			//get new price
			normalRandom=(1/sqrt(2.0*u01(rng)))*cos(2*PI*u01(rng));
			price+=price*normalRandom*factor;
			//find default probability
			defProb=exp(time*hr)-exp((time+timeStep)*hr);
			//update discount
			discount*=1.0/exp(DISCOUNT*timeStep);
			time=time+timeStep;
			sumCVA+=defProb*discount*price;
		}

		// divide by N
		return sumCVA;
	}
};


float genPaths(float _factor,vector<counterParties>& _cp)
{
	thrust::device_vector<counterParties> dcp(_cp.begin(),_cp.end());

	float CVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA(0.2),0.0f,thrust::plus<float>());
	return CVA/NUM_SIMULATIONS;
}

int main(){
	cout<<"starting..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;
	vector<counterParties> cp(PARTIES_NUM);
	setupCounterparties(cp);
	allocateDeals(cp);
	cout<<"Parties setup complete; "<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;

	float factor=sqrt(VARIANCE)*(YEARS/float(NUM_TIMESTEPS));

	float average=genPaths(factor,cp);
	cout<<"average: "<<(average/NUM_SIMULATIONS)<<endl;

	cout<<"ending..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;

	return 0;
}


