#include "hip/hip_runtime.h"
#include "parameters.h"
#include "setup.h"
#include <vector>

//x6y
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>


using namespace std;

struct get_CVA : public thrust::unary_function<unsigned int,float>
{
	//hazard rate
	const float hr;

	//initialize
	get_CVA(float _hr):hr(_hr){}

	__host__ __device__
	float operator()(unsigned int seed)
	{
		float sumCVA = 0;
		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		thrust::uniform_real_distribution<float> u01(0,1);

		float timeStep=YEARS/float(NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		double price=STARTING_PRICE;

		float factor=sqrt(VARIANCE)*(YEARS/float(NUM_TIMESTEPS));
		float normalRandom=0;
		;
		float discount=1;
		//run the required number of steps
		for(unsigned int i = 0; i < NUM_TIMESTEPS-1; ++i)
		{
			time=time+timeStep;
			//get new price
			normalRandom=(1/sqrt(2.0*u01(rng)))*cos(2*PI*u01(rng));
			price+=price*normalRandom*factor;
			//find default probability
			defProb=1.0f/exp((time-timeStep)*hr)-1.0f/exp(time*hr);
			//update discount
			discount=1.0/exp(DISCOUNT*time);

			sumCVA+=defProb*discount*price;
			//cout<<i<<" price: "<<price<<" disc: "<<discount<<" defProb: "<<defProb<<endl;

		}
		cout<<sumCVA<<endl;
		return sumCVA;
	}
};


float genPaths(vector<counterParties>& _cp)
{
	//thrust::device_vector<counterParties> dcp(_cp.begin(),_cp.end());

	float CVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA(0.2),0.0f,thrust::plus<float>());
	return CVA/NUM_SIMULATIONS;
}

int main(){
	cout<<"starting..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;
	vector<counterParties> cp(PARTIES_NUM);
	setupCounterparties(cp);
	allocateDeals(cp);
	cout<<"Parties setup complete; "<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;

	cout<<"average: "<<genPaths(cp)<<endl;

	cout<<"ending..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;

	return 0;
}


