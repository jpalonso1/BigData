#include "hip/hip_runtime.h"
//Juan Pablo Alonso

#include <string>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/device_malloc.h>

#include "parameters.h"
#include "setup.h"
#include "xlog.h"

using namespace std;

struct counterpartyCVA
{
	//DESC: holds the normalized simulation results for each type of counterparty
	float normalizedCVA[5];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){normalizedCVA[i]=0;}
	}
};

__host__ __device__
counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR){
	//DESC: operator to be called in thrust binary operation to aggregate the results of each simulation
	//IN: two counterpartyCVA object to be added
	//OUT: a counterpartyCVA object containing the sum of both input objects
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++){
		tempCVA.normalizedCVA[i]=cvaL.normalizedCVA[i]+cvaR.normalizedCVA[i];
	}
	return tempCVA;
}

struct get_CVA : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	paramStruct pard;
	get_CVA(paramStruct _pard):pard(_pard){}

	__host__ __device__
	counterpartyCVA operator()(unsigned int seed)
	{
		//DESC: functor to run each simulation
		//IN: seed for random number generation
		//OUT: CVA object to be aggregated

		//initialize output counterparty results
		counterpartyCVA sumCVA;

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		thrust::random::experimental::normal_distribution<float> ndist(0, 1.0f);

		//initialize parameters for simulation
		float timeStep=pard.YEARS/float(pard.NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		float price=pard.STARTING_PRICE;
		float discount=1;
		//factor used in random evolution of price
		float priceFactor=sqrt(pard.VARIANCE)*(timeStep);

		//to hold the random normal generated each step
		float normal=ndist(rng);

		//initialize hazard rate factors (TO BE PARAMETRIZED?
		float hazard[5];
		for (int i=0;i<5;i++){
			hazard[i]=pard.BASE_HAZARD+pard.BASE_HAZARD*float(i);
		}

		//run the required number of steps
		//NOTE: TO BE OPTIMIZED
		for(unsigned int i = 0; i < pard.NUM_TIMESTEPS-1; ++i){
			time=time+timeStep;
			//get new price

			normal=ndist(rng);
			price+=price*normal*priceFactor;
			//get discount for current step
			discount=1.0/exp(pard.DISCOUNT*time);
			//find default probability for each and copy result to output CVA struct
			for (int j=0;j<5;j++){
				defProb=1.0f/exp((time-timeStep)*hazard[j])-1.0f/exp(time*hazard[j]);
				sumCVA.normalizedCVA[j]+=defProb*discount*price;
			}
			normal=ndist(rng);
		}
		return sumCVA;
	}
};

counterpartyCVA genPaths();
float getCumulativeCVA(counterpartyCVA& cpCVA,vector<counterParties>& cp);

int main(){
	XLog logMain("CVA Main");

	//-----------------Setup
	//initialize counterparties vector
	XLog logAlloc("Setup");
	vector<counterParties> cp(parh.PARTIES_NUM);
	//intialize counterparties CVA
	setupCounterparties(cp);
	//assign deals to counterparties randomly based on ratio
	allocateDeals(cp);
	logAlloc.end();

	//----------------Simulation
	XLog logPath("Path simulation");
	counterpartyCVA cpCVA;
	cpCVA=genPaths();
	logPath.end();

	//---------------Aggregation
	XLog logSum("Aggregate CVA");
	float totalCVA;
	totalCVA=getCumulativeCVA(cpCVA,cp);
	logSum.log("total CVA:",totalCVA);

	logMain.end();
	return 0;
}

counterpartyCVA genPaths(){
	//DESC: simulates the CVA and obtains the average
	//OUT: Counterparty factor object

	//CVA aggregator for simulations run
    thrust::plus<counterpartyCVA> binary_op;
    counterpartyCVA cpCVA;
    //run the simulation using thrust reduction
    cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(parh.NUM_SIMULATIONS),get_CVA(parh),cpCVA,binary_op);

	for (int i=0;i<5;i++)
	{cpCVA.normalizedCVA[i]=cpCVA.normalizedCVA[i]/float(parh.NUM_SIMULATIONS);}
	return cpCVA;
}

float getCumulativeCVA(counterpartyCVA& cpCVA,vector<counterParties>& cp){
	//DESC: uses the normalized CVA factors to get the total CVA for each counterparty
	//and aggregates it
	//IN: normalized CVA factor, vector containing all counterParties
	float sumCVA=0;
	int partiesFifth = parh.PARTIES_NUM / 5;
	for (int j = 0; j < 5; j++) {
		int startCount = partiesFifth * j;
		for (long i = 0; i < partiesFifth; i++) {
			sumCVA+=cpCVA.normalizedCVA[j]*cp[startCount + i].netDeal;
		}
	}
	return sumCVA;
}
