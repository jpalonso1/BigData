#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/device_malloc.h>

#include "parameters.h"
#include "setup.h"
#include "xlog.h"

using namespace std;

//holds properties/parameters updated from parameter file
const paramStruct parh=initParameters();

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCVA[5];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){normalizedCVA[i]=0;}
	}
};

//operator to be called in thrust binary operation
__host__ __device__
counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR)
{
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++)
	{
		tempCVA.normalizedCVA[i]=cvaL.normalizedCVA[i]+cvaR.normalizedCVA[i];
	}
	return tempCVA;
}

struct get_CVA : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	paramStruct par;
	get_CVA(paramStruct _par):par(_par){}

	__host__ __device__
	counterpartyCVA operator()(unsigned int seed)
	{
		//initialize output counterparty results
		counterpartyCVA sumCVA;

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		thrust::random::experimental::normal_distribution<float> ndist(0, 1.0f);

		//initialize parameters for simulation
		float timeStep=YEARS/float(par.NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		float price=par.STARTING_PRICE;
		float discount=1;
		//factor used in random evolution of price
		float priceFactor=sqrt(par.VARIANCE)*(timeStep);

		//to hold the random normal generated each step
		float normal=ndist(rng);

		//initialize hazard rate factors (TO BE PARAMETRIZED?
		float hazard[5];
		for (int i=0;i<5;i++)
		{
			hazard[i]=par.BASE_HAZARD+par.BASE_HAZARD*float(i);
		}

		//run the required number of steps
		//NOTE: TO BE OPTIMIZED
		for(unsigned int i = 0; i < par.NUM_TIMESTEPS-1; ++i)
		{
			time=time+timeStep;
			//get new price

			normal=ndist(rng);
			price+=price*normal*priceFactor;
			//get discount for current step
			discount=1.0/exp(par.DISCOUNT*time);
			//find default probability for each and copy result to output CVA struct
			for (int j=0;j<5;j++)
			{
				defProb=1.0f/exp((time-timeStep)*hazard[j])-1.0f/exp(time*hazard[j]);
				sumCVA.normalizedCVA[j]+=defProb*discount*price;
			}
			normal=ndist(rng);
		}
		return sumCVA;
	}
};

counterpartyCVA genPaths()
{
//	//update parameters from parameter file
//	paramStruct parh;
//	parh=initParameters();

    thrust::plus<counterpartyCVA> binary_op;
    counterpartyCVA cpCVA;
    XLog logInTr("Inside Transform");
    logInTr.start();
	cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA(parh),cpCVA,binary_op);
	logInTr.end();
	cout<<"Transform end"<<endl;
	for (int i=0;i<5;i++)
	{cpCVA.normalizedCVA[i]=cpCVA.normalizedCVA[i]/float(NUM_SIMULATIONS);}
	return cpCVA;
}

float getCumulativeCVA(counterpartyCVA& cpCVA,vector<counterParties>& cp)
{
	float sumCVA=0;
	int partiesFifth = PARTIES_NUM / 5;
	for (int j = 0; j < 5; j++) {
		int startCount = partiesFifth * j;
		for (long i = 0; i < partiesFifth; i++) {
//			cout<<cp[startCount + i].netDeal<<endl;
			sumCVA+=cpCVA.normalizedCVA[j]*cp[startCount + i].netDeal;
		}
	}
	return sumCVA;
}

int main(){
	XLog logMain("CVA Main");
	logMain.start();


	//-----------------Setup-----------------------
	//initialize counterparties vector
	XLog logAlloc("Setup");
	vector<counterParties> cp(PARTIES_NUM);
	//intialize counterparties CVA
	setupCounterparties(cp);
	logAlloc.log("Counterparties Setup");
	//assign deals to counterparties randomly based on ratio
	allocateDeals(cp);
	logAlloc.log("Deal allocation complete");
	logAlloc.end();

	counterpartyCVA cpCVA;
	XLog logPath("Path simulation");
	cpCVA=genPaths();
	logPath.end();

	float totalCVA;
	{
		XLog logSum("Sum CVA");
		totalCVA=getCumulativeCVA(cpCVA,cp);
		logSum.log("total CVA:",totalCVA);
	}
	logMain.end();
	return 0;
}
