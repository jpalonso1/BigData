#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/device_malloc.h>

#include "parameters.h"
#include "setup.h"
#include "xlog.h"

using namespace std;

const XParams param("paramhw2.txt");

paramStruct initParameters(){
	paramStruct tempPar;
	tempPar.NUM_SIMULATIONS= param.getLong("NUM_SIMULATIONS",10000);
	tempPar.NUM_TIMESTEPS= param.getLong("NUM_TIMESTEPS",1000);
	return tempPar;
}

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCVA[5];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){normalizedCVA[i]=0;}
	}
};

//operator to be called in thrust binary operation
__host__ __device__
counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR)
{
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++)
	{
		tempCVA.normalizedCVA[i]=cvaL.normalizedCVA[i]+cvaR.normalizedCVA[i];
	}
	return tempCVA;
}

struct get_CVA : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	thrust::device_ptr<paramStruct> raw_par;
	get_CVA(thrust::device_ptr<paramStruct> _raw_par):raw_par(_raw_par){}

	__host__ __device__
	counterpartyCVA operator()(unsigned int seed)
	{
		paramStruct * par = thrust::raw_pointer_cast(raw_par);
//		paramStruct * raw_ptr = thrust::raw_pointer_cast(pardi);

		//initialize output counterparty results
		counterpartyCVA sumCVA;

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		//thrust::uniform_real_distribution<float> u01(0,1);
		thrust::random::experimental::normal_distribution<float> ndist(0, 1.0f);

		//initialize parameters for simulation
		float timeStep=YEARS/float(NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		double price=STARTING_PRICE;
		float discount=1;

		//factor used in random evolution of price
		float priceFactor=sqrt(VARIANCE)*(YEARS/float(NUM_TIMESTEPS));

		//to hold the random normal generated each step

		float normal=0;

		//initialize hazard rate factors (TO BE PARAMETRIZED?
		float hazard[5];
		for (int i=0;i<5;i++)
		{
			hazard[i]=BASE_HAZARD+BASE_HAZARD*float(i);
		}

		//run the required number of steps
		//NOTE: TO BE OPTIMIZED
		for(unsigned int i = 0; i < NUM_TIMESTEPS-1; ++i)
		{
			time=time+timeStep;
			//get new price
			normal=ndist(rng);
			price+=price*normal*priceFactor;
			//get discount for current step
			discount=1.0/exp(DISCOUNT*time);
			//find default probability for each and copy result to output CVA struct
			for (int j=0;j<5;j++)
			{
				defProb=1.0f/exp((time-timeStep)*hazard[j])-1.0f/exp(time*hazard[j]);
//				cout<<j<<" defprob: "<<defProb<<" discount: "<<discount<<" price: "<<price<<endl;
				sumCVA.normalizedCVA[j]+=defProb*discount*price;
//				cout<<i<<" type: "<<j<<" CVA norm: "<<(defProb*discount*price)<<endl;
			}
		}
		return sumCVA;
	}
};

counterpartyCVA genPaths()
{
	paramStruct parh;
	parh=initParameters();

	thrust::device_ptr<paramStruct> dev_ptr = thrust::device_malloc<paramStruct>(1);
	dev_ptr[0]=parh;
//	paramStruct * par_ptr;
//	hipMalloc((void **) &par_ptr,sizeof(parh));
//	hipMemcpy(par_ptr,&parh,sizeof(parh), hipMemcpyHostToDevice);
//	thrust::device_ptr<paramStruct> dev_ptr(par_ptr);

	thrust::plus<counterpartyCVA> binary_op;
	counterpartyCVA cpCVA;
	cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA(dev_ptr),cpCVA,binary_op);
	for (int i=0;i<5;i++)
	{cpCVA.normalizedCVA[i]=cpCVA.normalizedCVA[i]/float(NUM_SIMULATIONS);}
	return cpCVA;
}

float getCumulativeCVA(counterpartyCVA& cpCVA,vector<counterParties>& cp)
{
	float sumCVA=0;
	int partiesFifth = PARTIES_NUM / 5;
	for (int j = 0; j < 5; j++) {
		int startCount = partiesFifth * j;
		for (long i = 0; i < partiesFifth; i++) {
			sumCVA+=cpCVA.normalizedCVA[j]*cp[startCount + i].netDeal;
		}
	}
	return sumCVA;
}

int main(){
	XLog logMain("CVA Main");
	logMain.start();


//	paramStruct* pard;
//	hipMalloc((void**)&pard,sizeof(parh));
//	hipMemcpy(pard,&parh,sizeof(parh),hipMemcpyHostToDevice);
//	thrust::device_ptr<paramStruct> dev_ptr2(pard);
//	dev_ptr=dev_ptr2;

//	thrust::device_ptr<paramStruct> dev_ptr = thrust::device_malloc<paramStruct>(1);
//    paramStruct* pard=thrust::raw_pointer_cast(dev_ptr);

//	cout<<"NUM_TIMESTEPS: "<<parh.NUM_TIMESTEPS<<endl;
	vector<counterParties> cp(PARTIES_NUM);
	{
		XLog logAlloc("Setup");
		setupCounterparties(cp);
		logAlloc.log("Counterparties Setup");
		allocateDeals(cp);
		logAlloc.log("Deal allocation complete");
	}

	counterpartyCVA cpCVA;
	{
		XLog logPath("Path simulation");
		cpCVA=genPaths();
		logPath.end();
	}

	float totalCVA;
	{
		XLog logSum("Sum CVA");
		totalCVA=getCumulativeCVA(cpCVA,cp);
		logSum.log("total CVA:",totalCVA);
	}
	logMain.end();
	return 0;
}
