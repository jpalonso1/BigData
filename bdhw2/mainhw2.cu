#include "hip/hip_runtime.h"
#include "parameters.h"
#include "setup.h"
#include <vector>

//x6y
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>


using namespace std;

//holds the normalized simulation results for each type of counterpary
struct counterpartyCVA
{
	float normalizedCVA[5];

//	counterpartyCVA operator+(const counterpartyCVA &cvaR)
//	{
//		counterpartyCVA tempCVA;
//		for(int i=0;i<5;i++)
//		{
//			tempCVA.normalizedCVA[i]=normalizedCVA[i]+cvaR.normalizedCVA[i];
//		}
//		return tempCVA;
//	}

};

counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR)
{
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++)
	{
		tempCVA.normalizedCVA[i]=cvaL.normalizedCVA[i]+cvaR.normalizedCVA[i];
	}
	return tempCVA;
}

struct get_CVA : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	//hazard rate
	const float hr;

	//initialize
	get_CVA(float _hr):hr(_hr){}

	__host__ __device__
	counterpartyCVA operator()(unsigned int seed)
	{
		//intialize counterparties and set to 0
		counterpartyCVA sumCVA;
		for (int i=0;i<5;i++)
		{
			sumCVA.normalizedCVA[i]=0;
		}
		// seed a random number generator
		thrust::default_random_engine rng(seed);

		// create a mapping from random numbers to [0,1)
		thrust::uniform_real_distribution<float> u01(0,1);

		float timeStep=YEARS/float(NUM_TIMESTEPS);
		float time=0;
		float defProb=0;
		double price=STARTING_PRICE;

		float factor=sqrt(VARIANCE)*(YEARS/float(NUM_TIMESTEPS));
		float normalRandom=0;
		;
		float discount=1;
		//run the required number of steps
		for(unsigned int i = 0; i < NUM_TIMESTEPS-1; ++i)
		{
			time=time+timeStep;
			//get new price
			normalRandom=(1/sqrt(2.0*u01(rng)))*cos(2*PI*u01(rng));
			price+=price*normalRandom*factor;
			//find default probability
			defProb=1.0f/exp((time-timeStep)*hr)-1.0f/exp(time*hr);
			//update discount
			discount=1.0/exp(DISCOUNT*time);

			sumCVA.normalizedCVA[0]+=defProb*discount*price;
		}
		return sumCVA;
	}
};


float genPaths(vector<counterParties>& _cp)
{
	thrust::plus<counterpartyCVA> binary_op;
	counterpartyCVA cpCVA;
	cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA(0.2),cpCVA,binary_op);
	return cpCVA.normalizedCVA[0]/NUM_SIMULATIONS;
}

int main(){
	cout<<"starting..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;
	vector<counterParties> cp(PARTIES_NUM);
	setupCounterparties(cp);
	allocateDeals(cp);
	cout<<"Parties setup complete; "<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;

	cout<<"average: "<<genPaths(cp)<<endl;

	cout<<"ending..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;

	return 0;
}


