#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include "parameters4.h"
#include "setup4.h"
#include "xlog.h"
//x6y

using namespace std;

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCVA[5];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){normalizedCVA[i]=0;}
	}
};

int main(){
	XLog logMain("CVA 2 Main");
	logMain.log("Starting..");
	vector<counterParties> cp(PARTIES_NUM);
	{
		XLog logAlloc("Setup");
		setupCounterparties(cp);
		logAlloc.log("Counterparties creation complete");
		allocateDeals(cp);
		logAlloc.log("Deal allocation complete");
		string cpFile("counterparties.txt");
		writeCounterparties(cp,cpFile);
		logAlloc.log("Output file");
	}
	cout<<"ending..."<<float(clock()) / float(CLOCKS_PER_SEC)<<endl;
	return 0;
}


