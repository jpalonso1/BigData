#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include "parameters4.h"
#include "setup4.h"
#include "xlog.h"
//x6y

using namespace std;

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCVA[5];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){normalizedCVA[i]=0;}
	}
};

int main(){
	XLog logMain("CVA 2 Main");
	const long cpGroups=iMAX_CP_GROUP/PARTIES_NUM;
	counterParties cp[iMAX_CP_GROUP];
	{
		XLog logAlloc("Setup");
		logAlloc.start();
		setupCounterparties(cp, iMAX_CP_GROUP);
		logAlloc.log("Counterparties creation complete");
		allocateDeals(cp,iMAX_CP_GROUP);
		logAlloc.log("Deal allocation complete");
		string cpFile("counterparties.txt");
//		writeCounterparties(cp,cpFile);
		logAlloc.log("Output file");
		logAlloc.end();
		printCPDetails(cp[7]);
	}
	logMain.end();
	return 0;
}


