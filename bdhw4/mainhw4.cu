#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include "parameters4.h"
#include "setup4.h"
#include "xlog.h"

#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/random/normal_distribution.h>

using namespace std;

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCashCVA[5];
	float normalizedSwapFloatCVA[5][MAX_PERIODS];
	float normalizedSwapFixedCVA[5][MAX_PERIODS];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){
			normalizedCashCVA[i]=0;
			for (long j=0;j<MAX_PERIODS;j++){
				normalizedSwapFloatCVA[i][j]=0;
				normalizedSwapFixedCVA[i][j]=0;
			}
		}
	}
};

//operator to be called in thrust binary operation
__host__ __device__
counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR)
{
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++)
	{
		tempCVA.normalizedCashCVA[i]=cvaL.normalizedCashCVA[i]+cvaR.normalizedCashCVA[i];
		for (long j=0;j<MAX_PERIODS;j++){
			tempCVA.normalizedSwapFloatCVA[i][j]=
					cvaL.normalizedSwapFloatCVA[i][j]+cvaR.normalizedSwapFloatCVA[i][j];
			tempCVA.normalizedSwapFixedCVA[i][j]=
					cvaL.normalizedSwapFixedCVA[i][j]+cvaR.normalizedSwapFixedCVA[i][j];
		}
	}
	return tempCVA;
}

__host__ __device__
inline float getNSCurve(float * BS,float t){
	//0=beta0, 1=beta1, 2=beta2, 3=lambda
	float tOverL=t/BS[3];
	return BS[0]+BS[1]*BS[3]*(1-exp(-tOverL))/t+BS[2]*BS[3]*((1-exp(-tOverL))/(t-exp(-tOverL)));
//	return b0+b1*exp(-tOverL)+b2*tOverL*exp(-tOverL);
}

struct get_CVA4 : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	paramStruct pard;
	get_CVA4(paramStruct _pard):pard(_pard){}

	__host__ __device__
	counterpartyCVA operator()(unsigned long seed)
	{
		//initialize output counterparty results
		counterpartyCVA sumCVA;

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		//Standard Normal distribution
		thrust::random::experimental::normal_distribution<float> ndist(0, 1.0f);

		//initialize parameters for simulation
		float timeStep=float(pard.YEARS)/float(pard.SWAP_PERIODS);
		float defProb=0;
		double price=pard.STARTING_PRICE;

		//factor used in random evolution of price
		float priceFactor=sqrt(pard.VARIANCE)*(timeStep);

		//to hold the random normal generated each step for asset
		float normal=0;
		//to hold normal for NS curve
		float normalNS=0;

		//initialize hazard rate factors
		float hazard[5];
		for (int i=0;i<5;i++)
		{
			hazard[i]=pard.BASE_HAZARD+pard.BASE_HAZARD*float(i);
		}

		//initialize nelson siegel factors
		float NS0[4];
		float NS1[4];
		thrust::random::experimental::normal_distribution<float> normNS[4];
		for (int i=0;i<4;i++){
			NS0[i]=pard.NS.xBar[i];
			//Normal distribution for siegel curve
			normNS[i]=thrust::random::experimental::normal_distribution<float> (pard.NS.xBar[i], 1.0f);
		}

		float time=0;
		float curveRate=0;
		float curveRateLast=0;
		float discount=1;

		float sqTimeStep=sqrt(timeStep);
		float stepDisc=0;
		//eliminate first random number
		normal=ndist(rng);
		//probability of default this and last period
		//run the required number of steps
		for(unsigned long i = 0; i < pard.SWAP_PERIODS-1; ++i){
			time=time+timeStep;
			//get new price
			normal=ndist(rng);
			price+=price*normal*priceFactor;
			//update NS curve factors
			for (int j=0;j<4;j++){
				//generate factors for current step using nelson siegel
				normalNS=normNS[j](rng);
				NS1[j]=pard.NS.alpha[j]*(pard.NS.xBar[j]-NS0[j])+pard.NS.sd[j]*sqTimeStep*normalNS;
				NS0[j]=NS1[j];
			}
			curveRate=getNSCurve(NS1,pard.YEARS-time);
			//fix nan values (in low-probability case that function explodes) assign last found value
			if (curveRate!=curveRate)curveRate=curveRateLast;
			//prevent values from exploding
			else if(curveRate<0||curveRate>1)curveRate=curveRateLast;
			curveRateLast=curveRate;

			//override for testing
			curveRate=0.06;

			stepDisc=exp(-timeStep*curveRate);
			discount=discount*stepDisc;
			//find default probability for each and copy result to output CVA struct
			for (int j=0;j<5;j++){
				defProb=1.0f/exp((time-timeStep)*hazard[j])-1.0f/exp(time*hazard[j]);
				sumCVA.normalizedCashCVA[j]+=defProb*discount*price;
				sumCVA.normalizedSwapFixedCVA[j][i]=-defProb*discount;
				sumCVA.normalizedSwapFloatCVA[j][i]=defProb*stepDisc*curveRate*1.0/12.0;
			}
		}
		return sumCVA;
	}
};

counterpartyCVA genPaths()
{
	thrust::plus<counterpartyCVA> binary_op;
	counterpartyCVA cpCVA;
	cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(parh.NUM_SIMULATIONS),get_CVA4(parh),cpCVA,binary_op);
	//find averages for the CVA
	for (int i=0;i<5;i++){
		cpCVA.normalizedCashCVA[i]=cpCVA.normalizedCashCVA[i]/float(parh.NUM_SIMULATIONS);
		for (long j=0;j<parh.SWAP_PERIODS;j++){
			cpCVA.normalizedSwapFixedCVA[i][j]=cpCVA.normalizedSwapFixedCVA[i][j]/float(parh.NUM_SIMULATIONS);
			cpCVA.normalizedSwapFloatCVA[i][j]=cpCVA.normalizedSwapFloatCVA[i][j]/float(parh.NUM_SIMULATIONS);
		}
	}
	return cpCVA;
}

float getAverageCVA(counterpartyCVA& cpCVA,counterParties* cp,long size)
{
	float cashCVA=0;
	float floatCVA=0;
	float fixedCVA=0;
	long partiesFifth = size / 5;
	for (int j = 0; j < 5; j++) {
		long startCount = partiesFifth * j;
		for (long i = 0; i < partiesFifth; i++) {
			cashCVA+=cpCVA.normalizedCashCVA[j]*cp[startCount + i].netCashDeal;
			for (long k=0;k<parh.SWAP_PERIODS;k++){
				fixedCVA+=cpCVA.normalizedSwapFixedCVA[j][k]*cp[startCount+i].swapFixed[k];
				floatCVA+=cpCVA.normalizedSwapFloatCVA[j][k]*cp[startCount+i].swapFloatNom[k];
			}
		}
	}
	cout<<"total cash: "<<cashCVA<<endl;
	cout<<"total float: "<<floatCVA<<endl;
	cout<<"total fixed: "<<fixedCVA<<endl;
	return cashCVA+floatCVA+fixedCVA;
}


int main(){
	XLog logMain("CVA 2 Main");
	logMain.start();
	//break processing into groups to manage memory
//	const long cpBatches=PARTIES_NUM/iMAX_CP_GROUP+bool(PARTIES_NUM%iMAX_CP_GROUP);
	cout<<"batches: "<<parh.CP_BATCHES<<endl;
	//track sum of CVA from all batches
	float sumCVA=0;
	//manage deal allocation
	for (int i=0;i<parh.CP_BATCHES;i++){
		//allocate memory for a single batch
		counterParties cp[iMAX_CP_GROUP];

		XLog logAlloc("Setup");
		cout<<"counterparties:"<<endl;
		setupCounterparties(cp);
		cout<<"deals:"<<endl;
		allocateDeals(cp);
		cout<<"counterparties"<<endl;
		logAlloc.end();

		XLog logTransform("Transform");
		cout<<"Transform: "<<endl;
		counterpartyCVA cpCVA=genPaths();
		logTransform.end();

		XLog logSum("Aggregate CVA");
		float totalCVA=getAverageCVA(cpCVA,cp,iMAX_CP_GROUP);
		sumCVA+=totalCVA;
		logSum.log("batch CVA:",totalCVA);

	}
	logMain.log("total CVA:",sumCVA);
	logMain.end();
	return 0;
}
