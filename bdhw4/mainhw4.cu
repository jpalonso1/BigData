#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include "parameters4.h"
#include "setup4.h"
#include "xlog.h"
//x6y

using namespace std;

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCashCVA[5];
	float normalizedSwapFloatCVA[5][SWAP_PERIODS];
	float normalizedSwapFixedCVA[5][SWAP_PERIODS];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){
			normalizedCashCVA[i]=0;
			for (int j=0;j<SWAP_PERIODS;j++){
				normalizedSwapFloatCVA[i][j]=0;
				normalizedSwapFixedCVA[i][j]=0;
			}
		}
	}
};



int main(){
	XLog logMain("CVA 2 Main");
	const long cpGroups=iMAX_CP_GROUP/PARTIES_NUM;
	counterParties cp[iMAX_CP_GROUP];
	{
		XLog logAlloc("Setup");
		logAlloc.start();
		setupCounterparties(cp, iMAX_CP_GROUP);
		logAlloc.log("Counterparties creation complete");
		allocateDeals(cp,iMAX_CP_GROUP);
		logAlloc.log("Deal allocation complete");
		string cpFile("counterparties.txt");
//		writeCounterparties(cp,cpFile);
		saveCP(cp,"testBin",iMAX_CP_GROUP);
		logAlloc.log("Output file");

		logAlloc.end();

		printCPDetails(cp[7]);
	}
	logMain.end();
	return 0;
}


