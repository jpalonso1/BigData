#include "hip/hip_runtime.h"
#include <string>
#include <vector>

#include "parameters4.h"
#include "setup4.h"
#include "xlog.h"

#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/random/normal_distribution.h>

using namespace std;

//holds the normalized simulation results for each type of counterparty
struct counterpartyCVA
{
	float normalizedCashCVA[5];
	float normalizedSwapFloatCVA[5][SWAP_PERIODS];
	float normalizedSwapFixedCVA[5][SWAP_PERIODS];
	//intialize counterparties and set to 0
	__host__ __device__
	counterpartyCVA()
	{
		for (int i=0;i<5;i++){
			normalizedCashCVA[i]=0;
			for (long j=0;j<SWAP_PERIODS;j++){
				normalizedSwapFloatCVA[i][j]=0;
				normalizedSwapFixedCVA[i][j]=0;
			}
		}
	}
};

//operator to be called in thrust binary operation
__host__ __device__
counterpartyCVA operator+(const counterpartyCVA &cvaL, const counterpartyCVA &cvaR)
{
	counterpartyCVA tempCVA;
	for(int i=0;i<5;i++)
	{
		tempCVA.normalizedCashCVA[i]=cvaL.normalizedCashCVA[i]+cvaR.normalizedCashCVA[i];
		for (long j=0;j<SWAP_PERIODS;j++){
			tempCVA.normalizedSwapFloatCVA[i][j]=
					cvaL.normalizedSwapFloatCVA[i][j]+cvaR.normalizedSwapFloatCVA[i][j];
			tempCVA.normalizedSwapFixedCVA[i][j]=
					cvaL.normalizedSwapFixedCVA[i][j]+cvaR.normalizedSwapFixedCVA[i][j];
		}
	}
	return tempCVA;
}

struct get_CVA4 : public thrust::unary_function<unsigned int,counterpartyCVA>
{
	__host__ __device__
	counterpartyCVA operator()(unsigned long seed)
	{
		//initialize output counterparty results
		counterpartyCVA sumCVA;

		// seed a random number generator
		thrust::default_random_engine rng(seed);

		//Standard Normal distribution
		thrust::random::experimental::normal_distribution<float> ndist(0, 1.0f);

		//Normal distribution for siegel curve
		thrust::random::experimental::normal_distribution<float> ndistns(DISCOUNT, 1.0f);

		//initialize parameters for simulation
		float timeStep=float(YEARS)/float(SWAP_PERIODS);
		float defProb=0;
		double price=STARTING_PRICE;

		//factor used in random evolution of price
		float priceFactor=sqrt(VARIANCE)*(timeStep);

		//to hold the random normal generated each step for asset
		float normal=0;
		//to hold normal for NS curve
		float normalNS=0;

		//initialize hazard rate factors
		float hazard[5];
		for (int i=0;i<5;i++)
		{
			hazard[i]=BASE_HAZARD+BASE_HAZARD*float(i);
		}

		float time=0;
		//used for nelson siegel
		float x0=DISCOUNT;
		float x1=DISCOUNT;
		float thisDisc=0;
		float discount=1;
		float rateSD=sqrt(RATE_VARIANCE);
		float sqTimeStep=sqrt(timeStep);
		float stepDisc=0;
		//eliminate first random number
		normal=ndist(rng);
		//probability of default this and last period
		//run the required number of steps
//		if(seed==6)cout<<"start price: "<<price<<endl;
		for(unsigned long i = 0; i < SWAP_PERIODS-1; ++i)
		{
			time=time+timeStep;
			//get new price
			normal=ndist(rng);
//			if(i==1)cout<<"seed: "<<seed<<" normal 1: "<<normal<<endl;
//			if(seed==7 && i==0)cout<<"price factor: "<<priceFactor<<endl;
			price+=price*normal*priceFactor;
			//generate discount for current step using nelson siegel
			normalNS=ndistns(rng);
//			if (seed==8)cout<<i<<",norm: "<<normalNS<<",timest: "<<timeStep<<",";
			x1=ALPHA*(DISCOUNT-x0)+rateSD*sqTimeStep*normalNS;
			x0=x1;
			stepDisc=exp(-timeStep*x1);
			discount=discount*stepDisc;
//			if(seed==8)cout<<i<<','<<price<<','<<x1<<','<<discount<<endl;
			//find default probability for each and copy result to output CVA struct
			for (int j=0;j<5;j++)
			{
				defProb=1.0f/exp((time-timeStep)*hazard[j])-1.0f/exp(time*hazard[j]);
//				cout<<j<<" defprob: "<<defProb<<" discount: "<<discount<<" price: "<<price<<endl;
				sumCVA.normalizedCashCVA[j]+=defProb*discount*price;
				sumCVA.normalizedSwapFixedCVA[j][i]=defProb*discount;
				sumCVA.normalizedSwapFloatCVA[j][i]=defProb*stepDisc*x1*1.0/12.0;
//				if(seed==10)cout<<i<<" j: "<<j<<","<<sumCVA.normalizedSwapFixedCVA[j][i]<<endl;

			}
		}
		return sumCVA;
	}
};

counterpartyCVA genPaths()
{
	thrust::plus<counterpartyCVA> binary_op;
	counterpartyCVA cpCVA;
	cpCVA = thrust::transform_reduce(thrust::counting_iterator<int>(0),
			thrust::counting_iterator<int>(NUM_SIMULATIONS),get_CVA4(),cpCVA,binary_op);
	//find averages for the CVA
	for (int i=0;i<5;i++){
		cpCVA.normalizedCashCVA[i]=cpCVA.normalizedCashCVA[i]/float(NUM_SIMULATIONS);
		for (long j=0;j<SWAP_PERIODS;j++){
			cpCVA.normalizedSwapFixedCVA[i][j]=cpCVA.normalizedSwapFixedCVA[i][j]/float(NUM_SIMULATIONS);
			cpCVA.normalizedSwapFloatCVA[i][j]=cpCVA.normalizedSwapFloatCVA[i][j]/float(NUM_SIMULATIONS);
		}
	}
	return cpCVA;
}

float getCumulativeCVA(counterpartyCVA& cpCVA,counterParties* cp,long size)
{
	float sumCVA=0;
	float cashCVA=0;
	float floatCVA=0;
	float fixedCVA=0;
	long partiesFifth = size / 5;
	for (int j = 0; j < 5; j++) {
		long startCount = partiesFifth * j;
		for (long i = 0; i < partiesFifth; i++) {
			cashCVA+=cpCVA.normalizedCashCVA[j]*cp[startCount + i].netCashDeal;
			for (long k=0;k<SWAP_PERIODS;k++){
				fixedCVA+=cpCVA.normalizedSwapFixedCVA[j][k]*cp[startCount+i].swapFixed[k];
				floatCVA+=cpCVA.normalizedSwapFloatCVA[j][k]*cp[startCount+i].swapFloatNom[k];
			}
		}
	}
	cout<<"sum cash: "<<cashCVA<<endl;
	cout<<"sum fixed: "<<fixedCVA<<endl;
	cout<<"sum float: "<<floatCVA<<endl;
	sumCVA=cashCVA+floatCVA+fixedCVA;
	return sumCVA;
}

int main(){
	XLog logMain("CVA 2 Main");
	logMain.start();
	//break processing into groups to manage memory
//	const long cpBatches=PARTIES_NUM/iMAX_CP_GROUP+bool(PARTIES_NUM%iMAX_CP_GROUP);
	cout<<"batches: "<<CP_BATCHES<<endl;
	//manage deal allocation
//	const long cpPerBatch=PARTIES_NUM/cpBatches;

	for (int i=0;i<CP_BATCHES;i++){
		//allocate memory for a single batch
		counterParties cp[CP_PER_BATCH];
		XLog logAlloc("Setup");
		logAlloc.start();
		setupCounterparties(cp);
		logAlloc.log("Counterparties creation complete");
		allocateDeals(cp);
		logAlloc.log("Deal allocation complete");
		string cpFile("counterparties.txt");
//		writeCounterparties(cp,cpFile);
		saveCP(cp,"testBin");
		XLog logTransform("Transform");
		logTransform.start();
		counterpartyCVA cpCVA=genPaths();
		logTransform.end();
		logAlloc.log("Output file");
		cout<<"test deals: "<<cp[4200].numSwaps<<endl;
		logAlloc.end();
		{
			XLog logSum("Sum CVA");
			float totalCVA=getCumulativeCVA(cpCVA,cp,CP_PER_BATCH);
			logSum.log("total CVA:",totalCVA);
		}
	}
	logMain.end();
	return 0;
}
