#include "hip/hip_runtime.h"
//Juan Pablo Alonso

#include <iostream>
#include <cstdlib>
#include <fstream>
#include <string>
#include <ctime>
#include <time.h>
#include <sys/time.h>
#include "xlog.h"
#include <cstring>
#include <thrust/device_vector.h>

const long DUPLICATE_ARRAY_SIZE=60;
const long GROUP_STRING_SIZE=1000;
const long NUM_GROUPS=10000;
const long MAX_LINE_LENGTH=50;

using namespace std;

__host__ __device__
inline bool findPriceVolNoise(const char* lineChecked)
{
	/* Looks for specific noise in the price or volume
	 */
	//parse the line being tested
//	cout<<"line checked: "<<lineChecked<<endl;
	bool firstComma=false;
	for (int i=23;i<GROUP_STRING_SIZE;i++){
		if (lineChecked[i]==','){
			//value is being tested after first comma
			if (firstComma==false){
				firstComma=true;
				//check excessive prices positive or negative
				if (lineChecked[i+1]=='-'){
					if (lineChecked[i+2]!='0')return true;
				}
				else if(lineChecked[i+1]!='0'){
					return true;
				}
				i=i+7;
			}
			//negative volume tested after second comma
			else if (firstComma==true){
				if (lineChecked[i+1]=='-')return true;
				else return false;
			}
		}
	}
	//no noise found
	return false;
}

__host__ __device__
inline bool findDatetimeNoise(const char* lineChecked){
	/* Looks for specific noise in the date or time
	 */
	//look for weekend date
	if (lineChecked[7]=='2'){
//		cout<<"date noise 2: "<<lineChecked<<endl;
		return true;
	}
	//look for 9 am record
	if (lineChecked[10]=='9')return true;
	//look for 17 pm record
	if (lineChecked[9]=='1'&&lineChecked[10]=='7')return true;

	//no noise found
	return false;
}

struct group_lines{
	char line[GROUP_STRING_SIZE][MAX_LINE_LENGTH];
};

struct group_bool{
	bool lineCheck[GROUP_STRING_SIZE];
	__host__ __device__
	group_bool()
	{
		for (long i=0;i<GROUP_STRING_SIZE;i++){lineCheck[i]=false;}
	}
};

struct find_noise
{
    __host__ __device__
    group_bool operator()(const group_lines& gs, const group_bool& gb) const {
		group_bool output_group_bool;
		//loop through each element of comparison array and check for match
		for (long i=0;i<GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE;i++)
		{
//			if(1==2)break;
			if (gs.line[i][0]=='x')break;
			else if (findPriceVolNoise(gs.line[i]))output_group_bool.lineCheck[i]=true;
			else if(findDatetimeNoise(gs.line[i]))output_group_bool.lineCheck[i]=true;
			else
			{
				bool singleDupeFound;
				//check for duplicates
				for (long j=0;j<DUPLICATE_ARRAY_SIZE;j++)
				{
					int lineCheck=i+j+1;
					singleDupeFound=true;
					//compare line i with line i+j+1
					for (int j=MAX_LINE_LENGTH-10;j>=0;j--)
					{
						//check each character
						if(gs.line[i][j]!=gs.line[lineCheck][j]){
							singleDupeFound=false;
							break;
						}
					}
					if (singleDupeFound==true){
						output_group_bool.lineCheck[i]=true;
						break;
					}
				}
			}
		}
		return output_group_bool;
	}
};

inline void customGetLine(group_lines& inGroup,ifstream& inStream, long& count){
	int i=0;
	char tempChar='x';
	while (1==1){
		tempChar=inStream.get();
		if (tempChar=='\n'){
			inGroup.line[count][i]='\0';
			break;
		}
		inGroup.line[count][i]=tempChar;
		i++;
		if (i==MAX_LINE_LENGTH-1)return;
	}
}

inline void customCopyChar(group_lines& cpyGroup,group_lines& pstGroup,long& cpyCount,long& pstCount){
	for (int i=0;i<MAX_LINE_LENGTH;i++){
		cpyGroup.line[pstCount][i]=cpyGroup.line[cpyCount][i];
	}
}

int main(int argc,char* argv[]){
	//check for file name
	if (argc<2)
	{
		cout<<"NO FILENAME PROVIDED";
		return 0;
	}
	//ready objects for input-output
	ifstream input(argv[1]);

	ofstream signal("signal.txt");
	ofstream noise("noise.txt");
	//check that the entire file has been processed
	XLog logClean("Find Noise");
	thrust::host_vector<group_bool> Hbool(NUM_GROUPS);
	thrust::host_vector<group_lines> Hline (NUM_GROUPS);
	long sectionProcessed=0;

	while(!input.eof())
	{
		long structsCount=0;
		long instr=0;
		long cpystr=0;
		//tracks the last line
		string lineChecked;
		//hold object being copied
		group_lines tempGroup[2];
		//switches between first and second temp group
		bool fg=false;
		XLog logRead("Read Data");
		//start reading the file while within vector capacity

		while (structsCount<NUM_GROUPS && !input.eof())
		{
			getline(input,lineChecked);
			lineChecked.copy(tempGroup[fg].line[instr],MAX_LINE_LENGTH);
//			lineChecked.copy(Hline[structsCount].line[instr],MAX_LINE_LENGTH);
//			customGetLine(Hline[structsCount],input,instr);
//			lineChecked=Hline[structsCount].line[instr];

			if(instr>(GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE-1))
			{
				//copy line within duplicate range to first lines in next group
				cpystr=instr-GROUP_STRING_SIZE+DUPLICATE_ARRAY_SIZE;
				lineChecked.copy(tempGroup[!fg].
						line[instr-GROUP_STRING_SIZE+DUPLICATE_ARRAY_SIZE],MAX_LINE_LENGTH);
//				lineChecked.copy(Hline[structsCount+1].
//						line[instr-GROUP_STRING_SIZE+DUPLICATE_ARRAY_SIZE],MAX_LINE_LENGTH);
//				customCopyChar(Hline[structsCount],Hline[structsCount+1],
//						instr,cpystr);
				//reset values to start new group
				if (instr==(GROUP_STRING_SIZE-1)){
					Hline[structsCount]=tempGroup[fg];
					fg=!fg;
					structsCount++;
					instr=DUPLICATE_ARRAY_SIZE-1;
					if (input.eof())break;
				}
			}
			//copy leftover struct and flag remainder
			if (input.eof()){
				for (int i=instr+1;i<GROUP_STRING_SIZE;i++){
					tempGroup[fg].line[i][0]='x';
				}
//				Hline[structsCount]=tempGroup[fg];
				structsCount++;
				break;
			}
			instr++;
		}
		logRead.end();

		cout<<"Z"<<Hline[0].line[1]<<endl;
		cout<<"X"<<Hline[0].line[GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE]<<endl;
		cout<<"X"<<Hline[0].line[GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE+1]<<endl;
		cout<<"Y"<<Hline[1].line[0]<<endl;
		cout<<"Y"<<Hline[1].line[1]<<endl;

		XLog logCopy("Copy");
		thrust::device_vector<group_bool> Dbool(Hbool);
		thrust::device_vector<group_lines> Dline(Hline);
		logCopy.end();

		XLog logTransform("Transform");
		thrust::transform(Dline.begin(), Dline.begin()+structsCount-1, Dbool.begin(), Dbool.begin(), find_noise());
		logTransform.end();

		XLog logBoolCopy("Copy bool");
		Hbool=Dbool;
		logBoolCopy.end();
		XLog logOutput("output to file");
		//copy to noise and signal files
		for (long i=0;i<structsCount-1;i++)
		{
			//ignore overlapping section (-DUPLICATE_ARRAY_SIZE)
			for (long j=0;j<GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE;j++)
			{
				if (Hbool[i].lineCheck[j]==true)noise<<Hline[i].line[j]<<'\n';
				else signal<<Hline[i].line[j]<<'\n';
			}
		}

		logOutput.end();
		//process "leftover" strings
		for (long j=0;j<instr;j++)
		{
			if (Hbool[structsCount-1].lineCheck[j]==true)noise<<Hline[structsCount-1].line[j]<<'\n';
			else signal<<Hline[structsCount-1].line[j]<<'\n';
		}
//		gsOut=Hstr[structsCount-1];
//		gbOut=Hbool[structsCount-1];
//		for (long j=0;j<instr-1;j++)
//		{
//			if (gbOut.lineCheck[j]==true)noise<<gsOut.line[j]<<'\n';
//			else signal<<gsOut.line[j]<<'\n';
//		}
//
//		//quick check of values
//		group_values gsTest=Hstr[4];
//		group_bool gbTest=Hbool[4];
//		cout<<gbTest.lineCheck[7]<<" "<<gsTest.line[7]<<endl;;
//
		//get total noise found (optional)
		long sum=0;
		for(long i=0;i<structsCount+1;i++)
		{
			for (long j=0;j<GROUP_STRING_SIZE;j++)
			{
				sum+=Hbool[i].lineCheck[j];
			}
		}

		sectionProcessed++;
		long linesProcessed=sectionProcessed*(GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE)*NUM_GROUPS;
		logClean.log("Processed lines up to: ",linesProcessed);
		logClean.log("noise found:",sum);
	}
	logClean.end();
	return 0;
}


