#include "hip/hip_runtime.h"
//Juan Pablo Alonso

#include <iostream>
#include <cstdlib>
#include <fstream>
#include <string>
#include <ctime>
#include <time.h>
#include <sys/time.h>
#include "xlog.h"
#include <cstring>
#include <thrust/device_vector.h>

const long DUPLICATE_ARRAY_SIZE=60;
const long GROUP_STRING_SIZE=2000;
const long NUM_GROUPS=10000;
const long MAX_LINE_LENGTH=50;

using namespace std;

ofstream signal;
ofstream noise;

struct group_lines{
	char line[GROUP_STRING_SIZE][MAX_LINE_LENGTH];
};

struct group_bool{
	bool lineCheck[GROUP_STRING_SIZE];
	__host__ __device__
	group_bool()
	{
		for (long i=0;i<GROUP_STRING_SIZE;i++){lineCheck[i]=false;}
	}
};

__host__
inline bool findPriceVolNoise(const char* lineChecked)
{
	/* Looks for specific noise in the price or volume
	 */
	//parse the line being tested
//	cout<<"line checked: "<<lineChecked<<endl;
	bool firstComma=false;
	for (int i=23;i<GROUP_STRING_SIZE;i++){
		if (lineChecked[i]==','){
			//value is being tested after first comma
			if (firstComma==false){
				firstComma=true;
				//check excessive prices positive or negative
				if (lineChecked[i+1]=='-'){
					if (lineChecked[i+2]!='0')return true;
				}
				else if(lineChecked[i+1]!='0'){
					return true;
				}
				i=i+7;
			}
			//negative volume tested after second comma
			else if (firstComma==true){
				if (lineChecked[i+1]=='-')return true;
				else return false;
			}
		}
	}
	//no noise found
	return false;
}

__host__
inline bool findDatetimeNoise(const char* lineChecked){
	/* Looks for specific noise in the date or time
	 */
	//look for weekend date
	if (lineChecked[7]=='2'){
//		cout<<"date noise 2: "<<lineChecked<<endl;
		return true;
	}
	//look for 9 am record
	if (lineChecked[10]=='9')return true;
	//look for 17 pm record
	if (lineChecked[9]=='1'&&lineChecked[10]=='7')return true;

	//no noise found
	return false;
}

struct find_noise
{
    __host__
    group_bool operator()(const group_lines& gs, const group_bool& gb) const {
		group_bool output_group_bool;
		//loop through each element of comparison array and check for match
		for (long i=0;i<GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE;i++)
		{
			if (gs.line[i][0]=='x')break;
			else if (findPriceVolNoise(gs.line[i]))output_group_bool.lineCheck[i]=true;
			else if(findDatetimeNoise(gs.line[i]))output_group_bool.lineCheck[i]=true;
			else
			{
				bool singleDupeFound;
				//check for duplicates
				for (long j=0;j<DUPLICATE_ARRAY_SIZE;j++)
				{
					int lineCheck=i+j+1;
					singleDupeFound=true;
					//compare line i with line i+j+1
					for (int j=MAX_LINE_LENGTH-10;j>=0;j--)
					{
						//check each character
						if(gs.line[i][j]!=gs.line[lineCheck][j]){
							singleDupeFound=false;
							break;
						}
					}
					if (singleDupeFound==true){
						output_group_bool.lineCheck[i]=true;
						break;
					}
				}
			}
			//output to file
			if (output_group_bool.lineCheck[i])noise<<gs.line[i]<<'\n';
			else signal<<gs.line[i]<<'\n';
		}
		return output_group_bool;
	}
};

inline long readData(thrust::host_vector<group_lines>& Hline,ifstream& input){
	//tracks the last line
	string lineChecked;
	//hold object being copied
	group_lines tempGroup[2];
	//switches between first and second temp group
	bool fg=false;
	long structsCount=0;
	long instr=0;
	long cpystr=0;
	size_t len=0;
	//start reading the file while within vector capacity
	while (structsCount<NUM_GROUPS)
	{
		getline(input,lineChecked);
		len=lineChecked.copy(tempGroup[fg].line[instr],MAX_LINE_LENGTH);
		tempGroup[fg].line[instr][len]='\0';
		if(instr>(GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE-1))
		{
			//copy line within duplicate range to first lines in next group
			cpystr=instr-GROUP_STRING_SIZE+DUPLICATE_ARRAY_SIZE;
			lineChecked.copy(tempGroup[!fg].line[cpystr],MAX_LINE_LENGTH);

			//reset values to start new group
			if (instr==(GROUP_STRING_SIZE-1)){
				Hline[structsCount]=tempGroup[fg];
				fg=!fg;
				structsCount++;
				instr=DUPLICATE_ARRAY_SIZE-1;
				if (input.eof())break;
			}
		}
		//copy leftover struct and flag remainder
		if (input.eof()){
			for (int i=instr;i<GROUP_STRING_SIZE;i++){
				tempGroup[fg].line[i][0]='x';
			}
			Hline[structsCount]=tempGroup[fg];
			structsCount++;
			break;
		}
		//move to next line
		instr++;
	}
	return structsCount;
}

int main(int argc,char* argv[]){
	//check for file name
	if (argc<2)
	{
		cout<<"NO FILENAME PROVIDED";
		return 0;
	}
	//ready objects for input-output
	ifstream input(argv[1]);

	//request detailed log (read and transform times)
	bool detailedLog=false;
	if (argc>2 && argv[2][0]=='d')detailedLog=true;

	signal.open("signal.txt");
	noise.open("noise.txt");

	//time and log the clean/output process
	XLog logClean("Scrub");
	logClean.start();

	long sectionProcessed=0;
	thrust::host_vector<group_bool> Hbool(NUM_GROUPS);
	thrust::host_vector<group_lines> Hline (NUM_GROUPS);

	//check that the entire file has been processed
	while(!input.eof())
	{
		XLog logRead("Read Data");
		//read data and store in hline vector
		long structsCount=readData(Hline,input);
		if(detailedLog)logRead.log("Batch time");

		XLog logTransform("Transform and Output");

		//separate into noise and signal and send to output files
		thrust::transform(Hline.begin(), Hline.begin()+structsCount, Hbool.begin(), Hbool.begin(), find_noise());
		if(detailedLog)logTransform.log("Batch time");

		sectionProcessed++;
		long linesProcessed=sectionProcessed*(GROUP_STRING_SIZE-DUPLICATE_ARRAY_SIZE)*NUM_GROUPS;
		logClean.log("Processed lines up to: ",linesProcessed);
	}

	logClean.end();
	return 0;
}


